#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018 John Tromp
 * Copyright (c) 2018 Jiri Vadura - photon
 * Copyright (C) 2018-2020 The Merit Foundation
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either vedit_refsion 3 of the License, or
 * (at your option) any later vedit_refsion.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * In addition, as a special exception, the copyright holders give
 * permission to link the code of portions of this program with the
 * Botan library under certain conditions as described in each
 * individual source file, and distribute linked combinations
 * including the two.
 *
 * You must obey the GNU General Public License in all respects for
 * all of the code used other than Botan. If you modify file(s) with
 * this exception, you may extend this exception to your version of the
 * file(s), but you are not obligated to do so. If you do not wish to do
 * so, delete this exception statement from your version. If you delete
 * this exception statement from all source files in the program, then
 * also delete it here.
 */

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "exceptions.h"
#include "merit/nvml/nvml.h"
#include "merit/termcolor/termcolor.hpp"
#include <xmmintrin.h>
#include <algorithm>
#include <stdio.h>
#include <stdint.h>
#include <atomic>
#include <vector>
#include <map>
#include <string>
#include <iostream>
#include <sstream>
#include <set>
#include <fstream>
#include <cassert>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <merit/nvml/nvml.h>
#include <memory>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

typedef uint8_t u8;
typedef uint16_t u16;
typedef uint32_t u32;
typedef uint64_t u64;

typedef u32 node_t;
typedef u64 nonce_t;

#define MAXDEVICES 16
#define MAXPATHLEN 8192
#define MAXPROOFLENGTH 84
#define BIGEPS 5 / 64
#define TRIMFRAC256 184
#define BKTGRAN 32

const int CTHREADS = 1024;

template <class offset_t, uint8_t EDGEBITS_IN, uint8_t XBITS_IN>
struct Params {
    // prepare params for algorithm
    const static u32 XBITS = XBITS_IN;
    const static u32 NX = 1 << XBITS;
    const static u32 EDGEBITS = EDGEBITS_IN;
    const static u32 NEDGES = (offset_t)1 << EDGEBITS;
    const static u32 EDGEMASK = NEDGES - 1;
    const static u32 NODEBITS = EDGEBITS + 1;
    const static offset_t NNODES = (offset_t)1 << NODEBITS;
    const static u32 NODEMASK = NNODES - 1;

    const static u32 IDXSHIFT = 10;
    const static u32 CUCKOO_SIZE = NNODES >> IDXSHIFT;
    const static u32 CUCKOO_MASK = CUCKOO_SIZE - 1;
    const static u32 KEYBITS = 64-NODEBITS;
    const static u64 KEYMASK = (1LL << KEYBITS) - 1;
    const static u64 MAXDRIFT = 1LL << (KEYBITS - IDXSHIFT);

    const static u32 GABLOCKS = 512;
    const static u32 GATPB = 64;
    const static u32 GBBLOCKS = 32*BKTGRAN;
    const static u32 GBTPB = 64;
    const static u32 TRBLOCKS = 4096;
    const static u16 TRTPB = 1024;
    const static u16 TLBLOCKS = 4096;
    const static u16 TLTPB = 1024;
    const static u16 RBLOCKS = 512;
    const static u16 RTPB = 256;
};

struct SipKeys
{
    unsigned long long k0;
    unsigned long long k1;
    unsigned long long k2;
    unsigned long long k3;
};

// const auto DUCK_SIZE_A = 130LL;
// const auto DUCK_SIZE_B = 85LL;
const auto DUCK_SIZE_A = 30LL;
const auto DUCK_SIZE_B = 20LL;

const auto DUCK_A_EDGES = (DUCK_SIZE_A * 1024LL);
const auto DUCK_A_EDGES_64 = (DUCK_A_EDGES * 64LL);

const auto DUCK_B_EDGES = (DUCK_SIZE_B * 1024LL);
const auto buffer_size = DUCK_SIZE_A * 1024 * 4096 * 8;
const auto buffer_size_2 = DUCK_SIZE_B * 1024 * 4096 * 8;
const auto indexes_size = 128 * 128 * 4;

const int BKTMASK4K = (4096-1);


#define ROTL(x,b) ( ((x) << (b)) | ( (x) >> (64 - (b))) )
#define SIPROUND \
    do { \
        v0 += v1; v2 += v3; v1 = ROTL(v1,13); \
        v3 = ROTL(v3,16); v1 ^= v0; v3 ^= v2; \
        v0 = ROTL(v0,32); v2 += v1; v0 += v3; \
        v1 = ROTL(v1,17);   v3 = ROTL(v3,21); \
        v1 ^= v2; v3 ^= v0; v2 = ROTL(v2,32); \
    } while(0)


template<int EDGEMASK>
__device__  node_t dipnode(
        const u64 v0i,
        const u64 v1i,
        const u64 v2i,
        const u64 v3i,
        const  nonce_t nce,
        const  u32 uorv) {

    u64 nonce = 2 * nce + uorv;
    u64 v0 = v0i, v1 = v1i, v2 = v2i, v3 = v3i ^ nonce;
    SIPROUND; SIPROUND;
    v0 ^= nonce;
    v2 ^= 0xff;
    SIPROUND; SIPROUND; SIPROUND; SIPROUND;
    return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

template<int EDGEMASK>
node_t host_dipnode(
        const u64 v0i,
        const u64 v1i,
        const u64 v2i,
        const u64 v3i,
        const  nonce_t nce,
        const  u32 uorv) {

    u64 nonce = 2 * nce + uorv;
    u64 v0 = v0i, v1 = v1i, v2 = v2i, v3 = v3i ^ nonce;
    SIPROUND; SIPROUND;
    v0 ^= nonce;
    v2 ^= 0xff;
    SIPROUND; SIPROUND; SIPROUND; SIPROUND;
    return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

__device__ ulonglong4 Pack4edges(
        const uint2 e1,
        const  uint2 e2,
        const  uint2 e3,
        const  uint2 e4)
{
    u64 r1 = (((u64)e1.y << 32) | ((u64)e1.x));
    u64 r2 = (((u64)e2.y << 32) | ((u64)e2.x));
    u64 r3 = (((u64)e3.y << 32) | ((u64)e3.x));
    u64 r4 = (((u64)e4.y << 32) | ((u64)e4.x));
    return make_ulonglong4(r1, r2, r3, r4);
}

    template <class P>
__global__  void FluffyRecovery(
        const u64 v0i,
        const u64 v1i,
        const u64 v2i,
        const u64 v3i,
        uint8_t proof_size,
        ulonglong4 * buffer,
        int * indexes,
        u64* recovery)
{
    const int gid = blockDim.x * blockIdx.x + threadIdx.x;
    const int lid = threadIdx.x;

    __shared__ u32 nonces[MAXPROOFLENGTH];

    if (lid < proof_size) nonces[lid] = 0;

    __syncthreads();

    for (int i = 0; i < 1024 * 4; i++)
    {
        u64 nonce = gid * (1024 * 4) + i;
        if(nonce > P::NEDGES){
            break;
        }

        u64 u = dipnode<P::EDGEMASK>(v0i, v1i, v2i, v3i, nonce, 0) << 1;
        u64 v = dipnode<P::EDGEMASK>(v0i, v1i, v2i, v3i, nonce, 1) << 1 | 1;

        u64 a = u | (v << 32);
        u64 b = v | (u << 32);

        for (int i = 0; i < proof_size; i++)
        {
            if ((recovery[i] == a) || (recovery[i] == b))
                nonces[i] = nonce;
        }
    }

    __syncthreads();

    if (lid < proof_size)
    {
        if (nonces[lid] > 0)
            indexes[lid] = nonces[lid];
    }
}


    template<class P>
__global__  void FluffySeed2A(
        const u64 v0i,
        const u64 v1i,
        const u64 v2i,
        const u64 v3i,
        ulonglong4 * __restrict__ buffer,
        int *  __restrict__ indexes)
{
    const int gid = blockDim.x * blockIdx.x + threadIdx.x;
    const int lid = threadIdx.x;

    __shared__ uint2 tmp[P::NX][16];
    __shared__ int counters[P::NX];

    counters[lid] = 0;

    __syncthreads();

    u64 nonce    = (u64)gid * P::NEDGES / (blockDim.x * gridDim.x);
    u64 endnonce = (u64)(gid +1) * P::NEDGES / (blockDim.x * gridDim.x);
    for (; nonce < endnonce; nonce++) {

        uint2 hash;

        hash.x = dipnode<P::EDGEMASK>(v0i, v1i, v2i, v3i, nonce, 0);
        hash.y = dipnode<P::EDGEMASK>(v0i, v1i, v2i, v3i, nonce, 1);

        int bucket = hash.x & (64 - 1);

        __syncthreads();

        int counter = min((int)(atomicAdd(counters + bucket, 1)), (int)14);

        tmp[bucket][counter] = hash;

        __syncthreads();

        {
            int idx = min(16, counters[lid]);

            if (idx >= 8) {
                int new_count = idx - 8;
                counters[lid] = new_count;

                {
                    int cnt = min(
                            static_cast<int>(atomicAdd(indexes + lid, 8)), 
                            static_cast<int>(DUCK_A_EDGES_64 - 8));

                    {
                        buffer[(lid * DUCK_A_EDGES_64 + cnt) / 4] =
                            Pack4edges(tmp[lid][0], tmp[lid][1], tmp[lid][2], tmp[lid][3]);

                        buffer[(lid * DUCK_A_EDGES_64 + cnt + 4) / 4] =
                            Pack4edges(tmp[lid][4], tmp[lid][5], tmp[lid][6], tmp[lid][7]);
                    }
                }

                for (int t = 0; t < new_count; t++) {
                    tmp[lid][t] = tmp[lid][t + 8];
                }

            }
        }
    }

    __syncthreads();

    {
        int idx = min(15, counters[lid]);

        if (idx >  0) {
            int cnt = min( (int)atomicAdd(indexes + lid, 4), (int)(DUCK_A_EDGES_64 - 4));

            buffer[(lid * DUCK_A_EDGES_64 + cnt) / 4] = Pack4edges(
                    tmp[lid][0],
                    idx > 1 ? tmp[lid][1] : make_uint2(0, 0),
                    idx > 2 ? tmp[lid][2] : make_uint2(0, 0),
                    idx > 3 ? tmp[lid][3] : make_uint2(0, 0));
        }

        if (idx > 4) {
            int cnt = min(
                    static_cast<int>(atomicAdd(indexes + lid, 4)),
                    static_cast<int>(DUCK_A_EDGES_64 - 4));

            buffer[(lid * DUCK_A_EDGES_64 + cnt) / 4] = Pack4edges(
                    tmp[lid][4],
                    idx > 5 ? tmp[lid][5] : make_uint2(0, 0),
                    idx > 6 ? tmp[lid][6] : make_uint2(0, 0),
                    idx > 7 ? tmp[lid][7] : make_uint2(0, 0));
        }
    }

}

__global__  void FluffySeed2B(
        const  uint2 *__restrict__  source,
        ulonglong4 *__restrict__  destination,
        const  int *__restrict__  src_indexes,
        int *__restrict__  dest_indexes,
        int start_block)
{
    const int lid = threadIdx.x;
    const int group = blockIdx.x;

    __shared__ uint2 tmp[64][15];
    __shared__ int counters[64];

    counters[lid] = 0;

    __syncthreads();

    const int offset_mem = start_block * DUCK_A_EDGES_64;
    const int my_bucket = group / BKTGRAN;
    const int micro_block_no = group % BKTGRAN;

    const int bucket_edges = min(
            src_indexes[my_bucket + start_block],
            (int)(DUCK_A_EDGES_64));

    const int micro_block_edge_count = DUCK_A_EDGES_64 / BKTGRAN;
    const int loops = micro_block_edge_count / 64;

    for (int i = 0; i < loops; i++)
    {
        int edge_index = (micro_block_no * micro_block_edge_count) + (64 * i) + lid;

        if (edge_index < bucket_edges)
        {
            uint2 edge = source[offset_mem + (my_bucket * DUCK_A_EDGES_64) + edge_index];

            if (edge.x == 0 && edge.y == 0) continue;

            int bucket = (edge.x >> 6) & (64 - 1);

            __syncthreads();

            int counter = min((int)(atomicAdd(counters + bucket, 1)), (int)15);

            tmp[bucket][counter] = edge;

            __syncthreads();

            int idx = min(16, counters[lid]);

            if (idx >= 8) {
                int new_count = (idx - 8);
                counters[lid] = new_count;

                {
                    int cnt = min(
                            (int)atomicAdd(dest_indexes + start_block * 64 + my_bucket * 64 + lid, 8), 
                            static_cast<int>(DUCK_A_EDGES - 8));

                    {
                        destination[((my_bucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4] =
                            Pack4edges(tmp[lid][0], tmp[lid][1], tmp[lid][2], tmp[lid][3]);
                        destination[((my_bucket * 64 + lid) * DUCK_A_EDGES + cnt + 4) / 4] =
                            Pack4edges(tmp[lid][4], tmp[lid][5], tmp[lid][6], tmp[lid][7]);
                    }
                }

                for (int t = 0; t < new_count; t++) {
                    tmp[lid][t] = tmp[lid][t + 8];
                }

            }
        }
    }

    __syncthreads();

    {
        int idx = min(16, counters[lid]);

        if (idx > 0)
        {
            int cnt = min(
                    (int)atomicAdd(dest_indexes + start_block * 64 + my_bucket * 64 + lid, 4), 
                    (int)(DUCK_A_EDGES - 4));

            destination[((my_bucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4] = Pack4edges(
                    tmp[lid][0],
                    idx > 1 ? tmp[lid][1] : make_uint2(0, 0),
                    idx > 2 ? tmp[lid][2] : make_uint2(0, 0),
                    idx > 3 ? tmp[lid][3] : make_uint2(0, 0));
        }
        if (idx > 4)
        {
            int cnt = min(
                    (int)atomicAdd(dest_indexes + start_block * 64 + my_bucket * 64 + lid, 4), 
                    (int)(DUCK_A_EDGES - 4));

            destination[((my_bucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4] = Pack4edges(
                    tmp[lid][4],
                    idx > 5 ? tmp[lid][5] : make_uint2(0, 0),
                    idx > 6 ? tmp[lid][6] : make_uint2(0, 0),
                    idx > 7 ? tmp[lid][7] : make_uint2(0, 0));
        }
    }
}

__device__ __forceinline__  void Increase2bCounter(u32 * ecounters, const int bucket)
{
    int word = bucket >> 5;
    unsigned char bit = bucket & 0x1F;
    u32 mask = 1 << bit;

    u32 old = atomicOr(ecounters + word, mask) & mask;

    if (old > 0)
        atomicOr(ecounters + word + 4096, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 * ecounters, const int bucket)
{
    int word = bucket >> 5;
    unsigned char bit = bucket & 0x1F;
    u32 mask = 1 << bit;

    return (ecounters[word + 4096] & mask) > 0;
}

    template<class P, int BKTINSIZE, int BKTOUTSIZE>
__global__   void FluffyRound(
        const uint2 *__restrict__  source,
        uint2 *__restrict__  destination,
        const int *__restrict__  src_indexes,
        int * __restrict__ dest_indexes)
{

    const int lid = threadIdx.x;
    const int group = blockIdx.x;

    __shared__ u32 ecounters[8*CTHREADS];

    const int edges_in_bucket = min(src_indexes[group], BKTINSIZE);
    const int loops = (edges_in_bucket + CTHREADS) / CTHREADS;

    ecounters[lid] = 0;
    ecounters[lid + CTHREADS] = 0;
    ecounters[lid + (CTHREADS * 2)] = 0;
    ecounters[lid + (CTHREADS * 3)] = 0;
    ecounters[lid + (CTHREADS * 4)] = 0;
    ecounters[lid + (CTHREADS * 5)] = 0;
    ecounters[lid + (CTHREADS * 6)] = 0;
    ecounters[lid + (CTHREADS * 7)] = 0;

    __syncthreads();

    for (int i = 0; i < loops; i++) {
        const int lindex = (i * CTHREADS) + lid;

        if (lindex < edges_in_bucket) {
            const int index = (BKTINSIZE * group) + lindex;

            uint2 edge = source[index];

            if (edge.x == 0 && edge.y == 0) continue;

            Increase2bCounter(ecounters, (edge.x & P::EDGEMASK) >> 12);
        }
    }

    __syncthreads();

    for (int i = 0; i < loops; i++) {
        const int lindex = (i * CTHREADS) + lid;

        if (lindex < edges_in_bucket) {
            const int index = (BKTINSIZE * group) + lindex;

            uint2 edge = source[index];

            if (edge.x == 0 && edge.y == 0) continue;

            if (Read2bCounter(ecounters, (edge.x & P::EDGEMASK) >> 12))
            {
                const int bucket = edge.y & BKTMASK4K;
                const int bkt_idx = min(
                        atomicAdd(dest_indexes + bucket, 1),
                        BKTOUTSIZE - 1);

                destination[(bucket * BKTOUTSIZE) + bkt_idx] =
                    make_uint2(edge.y, edge.x);
            }
        }
    }

}


__global__   void /*Magical*/FluffyTail/*Pony*/(
        const uint2 * source,
        uint2 * destination,
        const int * src_indexes,
        int * dest_indexes)
{
    const int lid = threadIdx.x;
    const int group = blockIdx.x;

    int my_edges = src_indexes[group];
    __shared__ int dest_idx;

    if (lid == 0) {
        dest_idx = atomicAdd(dest_indexes, my_edges);
    }

    __syncthreads();

    if (lid < my_edges) {
        destination[dest_idx + lid] = source[group * DUCK_B_EDGES + lid];
    }
}

std::vector<u64> buffer_h;
std::vector<int*> buffer_a;
std::vector<int*> buffer_b;
std::vector<int*> indexes_e;
std::vector<int*> indexes_e2;
std::vector<u64*> recovery;

    template <class P>
bool TrimEdges(
        const SipKeys& k,
        int* buffer_a,
        int* buffer_b,
        int* indexes_e,
        int* indexes_e2)
{
    FluffySeed2A<P> << < P::GABLOCKS, P::GATPB >> > (
            k.k0, k.k1, k.k2, k.k3,
            (ulonglong4 *)buffer_a,
            (int *)indexes_e2);

    hipDeviceSynchronize();

    FluffySeed2B << < P::GBBLOCKS, P::GBTPB >> > (
            (const uint2 *)buffer_a,
            (ulonglong4 *)buffer_b,
            (const int *)indexes_e2,
            (int *)indexes_e,
            0);

    hipMemcpy(
            buffer_a,
            buffer_b,
            buffer_size / 2,
            hipMemcpyDeviceToDevice);

    FluffySeed2B << < P::GBBLOCKS, P::GBTPB >> > (
            (const uint2 *)buffer_a,
            (ulonglong4 *)buffer_b,
            (const int *)indexes_e2,
            (int *)indexes_e,
            32);

    hipMemcpy(
            &((char *)buffer_a)[buffer_size / 2],
            buffer_b,
            buffer_size / 2, hipMemcpyDeviceToDevice);


    hipMemset(indexes_e2, 0, indexes_size);
    FluffyRound<P, DUCK_A_EDGES, DUCK_B_EDGES> << < P::TRBLOCKS, P::TRTPB >> > (
            (const uint2 *)buffer_a,
            (uint2 *)buffer_b,
            (const int *)indexes_e,
            (int *)indexes_e2);

    hipDeviceSynchronize();

    for (int i = 0; i < 80; i++)
    {
        hipMemset(indexes_e, 0, indexes_size);
        FluffyRound<P, DUCK_B_EDGES, DUCK_B_EDGES> << < P::TRBLOCKS, P::TRTPB >> > (
                (const uint2 *)buffer_b,
                (uint2 *)buffer_a,
                (const int *)indexes_e2,
                (int *)indexes_e);

        hipMemset(indexes_e2, 0, indexes_size);
        FluffyRound<P, DUCK_B_EDGES, DUCK_B_EDGES> << < P::TRBLOCKS, P::TRTPB >> > (
                (const uint2 *)buffer_a,
                (uint2 *)buffer_b,
                (const int *)indexes_e,
                (int *)indexes_e2);
    }

    hipMemset(indexes_e, 0, indexes_size);
    hipDeviceSynchronize();

    FluffyTail << < P::TLBLOCKS, P::TLTPB >> > (
            (const uint2 *)buffer_b,
            (uint2 *)buffer_a,
            (const int *)indexes_e2,
            (int *)indexes_e);

    return true;
}

template<class P>
class CuckooHash {
    public:
        std::vector<u64> cuckoo;

        CuckooHash() : cuckoo(P::CUCKOO_SIZE) {
        }

        void set(node_t u, node_t v) {
            u64 niew = (u64)u << P::NODEBITS | v;
            for (node_t ui = u >> P::IDXSHIFT; ui < P::CUCKOO_SIZE ; ui = (ui+1) & P::CUCKOO_MASK) {
                u64 old = cuckoo[ui];
                if (old == 0 || (old >> P::NODEBITS) == (u & P::KEYMASK)) {
                    cuckoo[ui] = niew;
                    return;
                }
            }
        }
        node_t operator[](node_t u) const {
            for (node_t ui = u >> P::IDXSHIFT; ui < P::CUCKOO_SIZE; ui = (ui+1) & P::CUCKOO_MASK) {
                u64 cu = cuckoo[ui];
                if (!cu)
                    return 0;
                if ((cu >> P::NODEBITS) == (u & P::KEYMASK)) {
                    assert(((ui - (u >> P::IDXSHIFT)) & P::CUCKOO_MASK) < P::MAXDRIFT);
                    return (node_t)(cu & P::NODEMASK);
                }
            }
        }
};

template<class P>
u32 Path(CuckooHash<P> &cuckoo, u32 u, u32 *us) {
    u32 nu, u0 = u;
    for (nu = 0; u; u = cuckoo[u]) {
        if (nu >= MAXPATHLEN) {
            while (nu-- && us[nu] != u) ;
            if (~nu) {
                printf("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
                exit(0);
            }
            printf("maximum path length exceeded\n");
            return 0; // happens once in a million runs or so; signal trouble
        }
        us[nu++] = u;
    }
    return nu;
}

using Edge = std::pair<node_t, node_t>;

/*
   This function remains here because it provides a clear algorithm how the
   nonces are recovered from the edges. The Solution function after this one
   executes the same algorithm except on the GPU which is much much faster.
 */
template <class P>
void SolutionSlow(
        const SipKeys& keys,
        std::set<uint32_t >& nonces,
        node_t* us, u32 nu,
        node_t* vs, u32 nv) {

    std::set<Edge> cycle;
    cycle.insert(Edge{*us, *vs});

    while (nu--) {
        Edge e{us[(nu+1)&~1], us[nu|1]};
        cycle.insert(e); // u's in even position; v's in odd
    }

    while (nv--) {
        Edge e{vs[nv|1], vs[(nv+1)&~1]};
        cycle.insert(e); // u's in odd position; v's in even
    }

    for (u64 nonce = 0; nonce < P::NEDGES; nonce++) {
        u64 u = host_dipnode<P::EDGEMASK>(keys.k0, keys.k1, keys.k2, keys.k3, nonce, 0) << 1;
        u64 v = host_dipnode<P::EDGEMASK>(keys.k0, keys.k1, keys.k2, keys.k3, nonce, 1) << 1 | 1;

        Edge e{u,v};

        if (cycle.find(e) != cycle.end()) {
            nonces.insert(nonce);
            cycle.erase(e);
        }
    }
}

template <class P>
void Solution(
        uint8_t proof_size,
        const SipKeys& keys,
        std::set<uint32_t >& nonces,
        node_t* us, u32 nu,
        node_t* vs, u32 nv,
        int* buffer_a,
        int* indexes_e2,
        u64* recovery) {

    u64 solution_edges[MAXPROOFLENGTH];
    u32 host_nonces[MAXPROOFLENGTH];

    int i = 0;
    solution_edges[i] = (u64)*us | ((u64)(*vs) << 32); 
    i++;
    while (nu--) {
        solution_edges[i] = (u64)us[(nu+1)&~1] | ((u64)us[nu|1] << 32);
        i++;
    }

    while (nv--) {
        solution_edges[i] = (u64)vs[nv|1] | ((u64)vs[(nv+1)&~1] << 32);
        i++;
    }

    assert(i == proof_size);
    hipMemcpy(recovery, solution_edges, proof_size * 8, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipMemset(indexes_e2, 0, indexes_size);
    FluffyRecovery<P> << < P::RBLOCKS, P::RTPB >> >(
            keys.k0, keys.k1, keys.k2, keys.k3,
            proof_size,
            (ulonglong4 *)buffer_a,
            (int *)indexes_e2,
            recovery);
    hipDeviceSynchronize();
    hipMemcpy(host_nonces, indexes_e2, proof_size * 8, hipMemcpyDeviceToHost);
    for(int j = 0; j < i; j++) {
        nonces.insert(host_nonces[j]);
    }
}

using Cycle = std::set<uint32_t>;
using Cycles = std::vector<Cycle>;

template <class P>
bool FindCycles(
        const SipKeys& keys,
        Cycles& cycles, 
        u64* edges,
        const u32 size,
        uint8_t proof_size,
        int* buffer_a,
        int* indexes_e2,
        u64* recovery) {

    assert(proof_size <= MAXPROOFLENGTH);

    CuckooHash<P> cuckoo;
    node_t us[MAXPATHLEN], vs[MAXPATHLEN];

    for (u32 i = 0; i < size; i++) {
        u32 uxyz = edges[i] >> 32;  u32 vxyz = edges[i] & 0xffffffff;
        const u32 u0 = uxyz << 1, v0 = (vxyz << 1) | 1;
        if (u0) {
            u32 nu = Path(cuckoo, u0, us), nv = Path(cuckoo, v0, vs);
            if (!nu-- || !nv--) {
                return false; // drop edge causing trouble
            }

            if (us[nu] == vs[nv]) {
                const u32 min = nu < nv ? nu : nv;
                for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
                const u32 len = nu + nv + 1;

                if (len == proof_size) {
                    Cycle cycle;
                    Solution<P>(
                            proof_size,
                            keys,
                            cycle,
                            us,
                            nu,
                            vs,
                            nv,
                            buffer_a,
                            indexes_e2,
                            recovery);
                    if (cycle.size() == proof_size) {
                        cycles.emplace_back(cycle);
                    }
                }
            } else if (nu < nv) {
                while (nu--) {
                    cuckoo.set(us[nu+1], us[nu]);
                }
                cuckoo.set(u0, v0);
            } else {
                while (nv--) {
                    cuckoo.set(vs[nv+1], vs[nv]);
                }
                cuckoo.set(v0, u0);
            }
        }
    }
    return !cycles.empty();
}

int CudaDevices()
{
    int count = 0;
    hipGetDeviceCount(&count);
    return count;
}

const size_t BUFFER_H_SIZE = 150000;
int SetupKernelBuffers() {
    const int count = CudaDevices();

    if(!buffer_a.empty()) {
        return count;
    }

    assert(buffer_h.empty());
    assert(buffer_a.empty());
    assert(buffer_b.empty());
    assert(indexes_e.empty());
    assert(indexes_e2.empty());
    assert(recovery.empty());

    buffer_h.resize(count * BUFFER_H_SIZE);
    buffer_a.resize(count, nullptr);
    buffer_b.resize(count, nullptr);
    indexes_e.resize(count, nullptr);
    indexes_e2.resize(count, nullptr);
    recovery.resize(count, nullptr);

    return count;
}

using Cycle = std::set<uint32_t>;

template <class offset_t, uint8_t EDGEBITS, uint8_t XBITS>
struct Run
{
    using P = Params<offset_t, EDGEBITS, XBITS>;

    bool operator()(
            Cycles& cycles,
            uint64_t sip_k0, uint64_t sip_k1,
            uint8_t proof_size,
            int device)
    {
        assert(device >= 0);
        assert(device < buffer_a.size());

        u32 host_a[256 * 256];

        size_t free_device_mem = 0;
        size_t total_device_mem = 0;

        std::ostringstream err_msg;

        hipError_t status = hipSetDevice(device);
        if (status != hipSuccess) {
            err_msg << "An error occurred while trying to set the CUDA device: ";
            err_msg << hipGetErrorString(status);
            throw CudaSetDeviceException(err_msg.str());
        }

        if(buffer_a[device] == nullptr) {
            hipMemGetInfo(&free_device_mem, &total_device_mem);

            status = hipMalloc((void**)&buffer_a[device], buffer_size);
            if (status != hipSuccess) {
                err_msg << "An error while allocating memory for buffer_a: ";
                err_msg << hipGetErrorString(status);

                throw CudaMemoryAllocationException(err_msg.str());
            }
        }

        if(buffer_b[device] == nullptr) {
            status = hipMalloc((void**)&buffer_b[device], buffer_size_2);
            if (status != hipSuccess) {
                err_msg << "An error while allocating memory for buffer_b: ";
                err_msg << hipGetErrorString(status);

                throw CudaMemoryAllocationException(err_msg.str());
            }
        }

        if(indexes_e[device] == nullptr) {
            status = hipMalloc((void**)&indexes_e[device], indexes_size);
            if (status != hipSuccess) {
                err_msg << "An error while allocating memory for indexes_e: ";
                err_msg << hipGetErrorString(status);

                throw CudaMemoryAllocationException(err_msg.str());
            }
        }

        if(indexes_e2[device] == nullptr) {
            status = hipMalloc((void**)&indexes_e2[device], indexes_size);
            if (status != hipSuccess) {
                err_msg << "An error while allocating memory for indexes_e2: ";
                err_msg << hipGetErrorString(status);

                throw CudaMemoryAllocationException(err_msg.str());
            }
        }

        if(recovery[device] == nullptr) {
            status = hipMalloc((void**)&recovery[device], proof_size*8);
            if (status != hipSuccess) {
                err_msg << "An error while allocating memory for recovery: ";
                err_msg << hipGetErrorString(status);

                throw CudaMemoryAllocationException(err_msg.str());
            }
        }

        SipKeys keys {
            sip_k0 ^ 0x736f6d6570736575ULL,
                   sip_k1 ^ 0x646f72616e646f6dULL,
                   sip_k0 ^ 0x6c7967656e657261ULL,
                   sip_k1 ^ 0x7465646279746573ULL
        };

        hipMemset(indexes_e[device], 0, indexes_size);
        hipMemset(indexes_e2[device], 0, indexes_size);

        hipDeviceSynchronize();

        if(!TrimEdges<P>(
                    keys,
                    buffer_a[device],
                    buffer_b[device],
                    indexes_e[device],
                    indexes_e2[device])) {
            return false;
        }

        hipMemcpy(host_a, indexes_e[device], 64 * 64 * 4, hipMemcpyDeviceToHost);

        int pos = host_a[0];

        if(pos > 0 && pos < 500000) {
            hipMemcpy(
                    &buffer_h[device*BUFFER_H_SIZE],
                    &((u64*)buffer_a[device])[0],
                    pos * 8,
                    hipMemcpyDeviceToHost);
        }

        hipDeviceSynchronize();

        if(pos > 0 && pos < 500000) {
            return FindCycles<P>(
                    keys,
                    cycles,
                    &buffer_h[device*BUFFER_H_SIZE],
                    pos,
                    proof_size,
                    buffer_a[device],
                    indexes_e2[device],
                    recovery[device]);
        }
        return false;
    }
};

size_t CudaGetFreeMemory(int device){
    size_t free, total;

    hipSetDevice(device);
    hipDeviceReset();

    hipMemGetInfo(&free, &total);
    return free;
}

namespace nvml = merit::nvml;

std::unique_ptr<nvml::nvml_handle, int (*)(nvml::nvml_handle *)> initNVML(){
    auto nvml = std::unique_ptr<nvml::nvml_handle, int (*)(nvml::nvml_handle *)>(nvml::nvml_create(), nvml::nvml_destroy);

    if (nvml == nullptr)
        std::cerr << termcolor::red << "Failed to initialize NVML" << termcolor::reset << std::endl;

    return nvml;
}


std::vector<merit::GPUInfo> GPUsInfo()
{
    std::vector<merit::GPUInfo> res{};

    // Initialize NVML library
    auto nvml = initNVML();

    nvml::nvmlDevice_t device;
    int devices = CudaDevices();

    for (int index = 0; index < devices; index ++) {
        merit::GPUInfo item{};
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, index);
        item.id = index;
        item.title = prop.name;
        item.total_memory = prop.totalGlobalMem;

        // Get device
        auto nvmlres = nvml->nvmlDeviceGetHandleByIndex(index, &device);
        if (nvml::NVML_SUCCESS != nvmlres)
            std::cerr << termcolor::red << "Failed to get handle for device " << index << " " << nvml->nvmlErrorString(nvmlres) << termcolor::reset << std::endl;

        // Temperature
        unsigned int temp;
        nvmlres = nvml->nvmlDeviceGetTemperature(device, 0, &temp);
        if (nvml::NVML_SUCCESS != nvmlres){
            std::cerr << termcolor::red << "Failed to get temperature of device" << index << " " << nvml->nvmlErrorString(nvmlres) << termcolor::reset << std::endl;
            item.temperature = -1;
        } else {
            item.temperature = temp;
        }

        // GPU cores and memory utilization
        nvml::nvmlUtilization_t gpuUtil;
        nvmlres = nvml->nvmlDeviceGetUtilizationRates(device, &gpuUtil);
        if (nvml::NVML_SUCCESS != nvmlres){
            std::cerr << termcolor::red << "Failed to get utilization of device " << index << " : " << nvml->nvmlErrorString(nvmlres) << termcolor::reset << std::endl;
            item.gpu_util = -1;
            item.memory_util = -1;
        } else {
            item.gpu_util = gpuUtil.gpu;
            item.memory_util = gpuUtil.memory;
        }

        // Fan speed
        unsigned int speed;
        nvmlres = nvml->nvmlDeviceGetFanSpeed(device, &speed);
        if (nvml::NVML_SUCCESS != nvmlres){
            std::cerr << termcolor::red << "Failed to get fan speed of device " <<  index <<  " : " << nvml->nvmlErrorString(nvmlres) << termcolor::reset << std::endl;
            item.fan_speed = -1;
        } else {
            item.fan_speed = speed;
        }

        // add device info to array
        res.push_back(item);
    }

    return res;
}


bool FindCyclesOnCudaDevice(
        uint64_t sip_k0, uint64_t sip_k1,
        uint8_t edgebits,
        uint8_t proof_size,
        Cycles& cycles,
        int device)
{
    switch (edgebits) {
        case 16:
            return Run<uint32_t, 16u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 17:
            return Run<uint32_t, 17u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 18:
            return Run<uint32_t, 18u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 19:
            return Run<uint32_t, 19u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 20:
            return Run<uint32_t, 20u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 21:
            return Run<uint32_t, 21u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 22:
            return Run<uint32_t, 22u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 23:
            return Run<uint32_t, 23u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 24:
            return Run<uint32_t, 24u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 25:
            return Run<uint32_t, 25u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 26:
            return Run<uint32_t, 26u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 27:
            return Run<uint32_t, 27u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 28:
            return Run<uint32_t, 28u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 29:
            return Run<uint32_t, 29u, 6u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 30:
            return Run<uint64_t, 30u, 8u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        case 31:
            return Run<uint64_t, 31u, 8u>{}(cycles, sip_k0, sip_k1, proof_size, device);
        default:
            std::stringstream e;
            e << __func__ << ": Edgebits equal to " << edgebits << " is not supported";
            throw std::runtime_error(e.str());
    }
}
